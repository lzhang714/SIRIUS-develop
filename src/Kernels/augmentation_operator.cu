#include "hip/hip_runtime.h"
// Copyright (c) 2013-2019 Anton Kozhevnikov, Thomas Schulthess
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that
// the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the
//    following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions
//    and the following disclaimer in the documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED
// WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
// PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
// OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

/** \file augmentation_operator.cu
 *
 *  \brief CUDA kernels to generate augmentation operator and its derivative.
 */

#include "../SDDK/GPU/cuda_common.hpp"
#include "../SDDK/GPU/acc_runtime.hpp"

__global__ void aug_op_pw_coeffs_gpu_kernel(int ngvec__, int const* gvec_shell__, int const* idx__, int idxmax__,
                                            acc_complex_double_t const* zilm__, int const* l_by_lm__, int lmmax__,
                                            double const* gc__, int ld0__, int ld1__,
                                            double const* gvec_rlm__, int ld2__,
                                            double const* ri_values__, int ld3__, int ld4__,
                                            double* q_pw__, int ld5__, double fourpi_omega__)

{
    int igloc = blockDim.x * blockIdx.x + threadIdx.x;
    int idx12 = blockIdx.y;
    int idxsh = gvec_shell__[igloc];

    if (igloc < ngvec__) {
        int lm1     = idx__[array2D_offset(0, idx12, 3)];
        int lm2     = idx__[array2D_offset(1, idx12, 3)];
        int idxrf12 = idx__[array2D_offset(2, idx12, 3)];

        acc_complex_double_t z = make_accDoubleComplex(0, 0);
        for (int lm = 0; lm < lmmax__; lm++) {
            double d = gvec_rlm__[array2D_offset(lm, igloc, ld2__)] *
                ri_values__[array3D_offset(idxrf12, l_by_lm__[lm], idxsh, ld3__, ld4__)] *
                gc__[array3D_offset(lm, lm2, lm1, ld0__, ld1__)];
            z.x += d * zilm__[lm].x;
            z.y -= d * zilm__[lm].y;
        }
        q_pw__[array2D_offset(idx12, 2 * igloc,     ld5__)] = z.x * fourpi_omega__;
        q_pw__[array2D_offset(idx12, 2 * igloc + 1, ld5__)] = z.y * fourpi_omega__;
    }
}

extern "C" void aug_op_pw_coeffs_gpu(int ngvec__, int const* gvec_shell__, int const* idx__, int idxmax__,
                                     acc_complex_double_t const* zilm__, int const* l_by_lm__, int lmmax__,
                                     double const* gc__, int ld0__, int ld1__,
                                     double const* gvec_rlm__, int ld2__,
                                     double const* ri_values__, int ld3__, int ld4__,
                                     double* q_pw__, int ld5__, double fourpi_omega__)
{
    dim3 grid_t(32);
    dim3 grid_b(num_blocks(ngvec__, grid_t.x), idxmax__);

    accLaunchKernel((aug_op_pw_coeffs_gpu_kernel), dim3(grid_b), dim3(grid_t), 0, 0,
        ngvec__, gvec_shell__, idx__, idxmax__, zilm__, l_by_lm__, lmmax__, gc__, ld0__, ld1__, gvec_rlm__, ld2__,
        ri_values__, ld3__, ld4__, q_pw__, ld5__, fourpi_omega__);
}

