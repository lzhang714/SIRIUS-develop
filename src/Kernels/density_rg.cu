#include "hip/hip_runtime.h"
// Copyright (c) 2013-2018 Anton Kozhevnikov, Thomas Schulthess
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that
// the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the
//    following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions
//    and the following disclaimer in the documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED
// WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
// PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
// OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

/** \file density_rg.cu
 *
 *  \brief CUDA kernel to update density on the regular FFT grid.
 */

#include "../SDDK/GPU/cuda_common.hpp"
#include "../SDDK/GPU/acc_runtime.hpp"

__global__ void update_density_rg_1_complex_gpu_kernel(int size__,
                                                       acc_complex_double_t const* psi_rg__,
                                                       double wt__,
                                                       double* density_rg__)
{
    int ir = blockIdx.x * blockDim.x + threadIdx.x;
    if (ir < size__) {
        acc_complex_double_t z = psi_rg__[ir];
        density_rg__[ir] += (z.x * z.x + z.y * z.y) * wt__;
    }
}

/* Update one density component from one complex wave-function */
extern "C" void update_density_rg_1_complex_gpu(int size__, 
                                                acc_complex_double_t const* psi_rg__, 
                                                double wt__, 
                                                double* density_rg__)
{
    //CUDA_timer t("update_density_rg_1_gpu");

    dim3 grid_t(64);
    dim3 grid_b(num_blocks(size__, grid_t.x));

    accLaunchKernel((update_density_rg_1_complex_gpu_kernel), dim3(grid_b), dim3(grid_t), 0, 0, 
        size__,
        psi_rg__,
        wt__,
        density_rg__
    );
}

__global__ void update_density_rg_1_real_gpu_kernel(int size__,
                                                    double const* psi_rg__,
                                                    double wt__,
                                                    double* density_rg__)
{
    int ir = blockIdx.x * blockDim.x + threadIdx.x;
    if (ir < size__) {
        double p = psi_rg__[ir];
        density_rg__[ir] += p * p * wt__;
    }
}

/* Update one density component from one real wave-function */
extern "C" void update_density_rg_1_real_gpu(int size__,
                                             double const* psi_rg__,
                                             double wt__, 
                                             double* density_rg__)
{
    //CUDA_timer t("update_density_rg_1_gpu");

    dim3 grid_t(64);
    dim3 grid_b(num_blocks(size__, grid_t.x));

    accLaunchKernel((update_density_rg_1_real_gpu_kernel), dim3(grid_b), dim3(grid_t), 0, 0, 
        size__,
        psi_rg__,
        wt__,
        density_rg__
    );
}

__global__ void update_density_rg_2_gpu_kernel(int size__,
                                               acc_complex_double_t const* psi_up_rg__,
                                               acc_complex_double_t const* psi_dn_rg__,
                                               double wt__,
                                               double* density_x_rg__,
                                               double* density_y_rg__)
{
    int ir = blockIdx.x * blockDim.x + threadIdx.x;
    if (ir < size__) {
        acc_complex_double_t z = accCmul(psi_up_rg__[ir], accConj(psi_dn_rg__[ir]));
        density_x_rg__[ir] += 2 * z.x * wt__;
        density_y_rg__[ir] -= 2 * z.y * wt__;
    }
}

/* Update off-diagonal density component in non-collinear case */
extern "C" void update_density_rg_2_gpu(int size__,
                                        acc_complex_double_t const* psi_up_rg__,
                                        acc_complex_double_t const* psi_dn_rg__,
                                        double wt__,
                                        double* density_x_rg__,
                                        double* density_y_rg__)
{
    //CUDA_timer t("update_density_rg_1_gpu");

    dim3 grid_t(64);
    dim3 grid_b(num_blocks(size__, grid_t.x));

    accLaunchKernel((update_density_rg_2_gpu_kernel), dim3(grid_b), dim3(grid_t), 0, 0,
        size__,
        psi_up_rg__,
        psi_dn_rg__,
        wt__,
        density_x_rg__,
        density_y_rg__
    );
}


