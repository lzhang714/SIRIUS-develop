#include "hip/hip_runtime.h"
// Copyright (c) 2013-2018 Anton Kozhevnikov, Thomas Schulthess
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that
// the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the
//    following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions
//    and the following disclaimer in the documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED
// WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
// PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
// OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

/** \file residuals_aux.cu
 *
 *  \brief CUDA kernel to compute wave-function residuals on GPUs.
 */

#include "../SDDK/GPU/cuda_common.hpp"
#include "../SDDK/GPU/acc_runtime.hpp"

__global__ void compute_residuals_gpu_kernel
(
    int const num_rows_loc__,
    double const* eval__,
    acc_complex_double_t const* hpsi__,
    acc_complex_double_t const* opsi__,
    acc_complex_double_t* res__
)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int ibnd = blockIdx.y;

    if (j < num_rows_loc__) {
        int k = array2D_offset(j, ibnd, num_rows_loc__);
        /* res = hpsi_j - e_j * opsi_j */
        res__[k] = accCsub(hpsi__[k], make_accDoubleComplex(opsi__[k].x * eval__[ibnd], opsi__[k].y * eval__[ibnd]));
    }
}

//== __global__ void compute_residuals_norm_gpu_kernel
//== (
//==     int num_gkvec_row,
//==     int* res_idx,
//==     acc_complex_double_t const* res,
//==     double* res_norm,
//==     int reduced,
//==     int mpi_rank
//== )
//== {
//==     int N = num_blocks(num_gkvec_row, blockDim.x);
//== 
//==     ACC_DYNAMIC_SHARED( char, sdata_ptr)
//==     double* sdata = (double*)&sdata_ptr[0];
//== 
//==     sdata[threadIdx.x] = 0.0;
//== 
//==     for (int n = 0; n < N; n++)
//==     {
//==         int igk = n * blockDim.x + threadIdx.x;
//==         if (igk < num_gkvec_row)
//==         {
//==             int k = array2D_offset(igk, blockIdx.x, num_gkvec_row);
//==             sdata[threadIdx.x] += res[k].x * res[k].x + res[k].y * res[k].y;
//==         }
//==     }
//==     __syncthreads();
//== 
//==     for (int s = 1; s < blockDim.x; s *= 2)
//==     {
//==         if (threadIdx.x % (2 * s) == 0) sdata[threadIdx.x] = sdata[threadIdx.x] + sdata[threadIdx.x + s];
//==         __syncthreads();
//==     }
//== 
//==     if (!reduced)
//==     {
//==         res_norm[res_idx[blockIdx.x]] = sdata[0];
//==     }
//==     else
//==     {
//==         if (mpi_rank == 0)
//==         {
//==             double x = res[array2D_offset(0, blockIdx.x, num_gkvec_row)].x;
//==             res_norm[res_idx[blockIdx.x]] = 2 * sdata[0] - x * x;
//==         }
//==         else
//==         {
//==             res_norm[res_idx[blockIdx.x]] = 2 * sdata[0];
//==         }
//==     }
//== }
//== 
//== extern "C" void residuals_aux_gpu(int num_gvec_loc__,
//==                                   int num_res_local__,
//==                                   int* res_idx__,
//==                                   double* eval__,
//==                                   acc_complex_double_t const* hpsi__,
//==                                   acc_complex_double_t const* opsi__,
//==                                   double const* h_diag__,
//==                                   double const* o_diag__,
//==                                   acc_complex_double_t* res__,
//==                                   double* res_norm__,
//==                                   double* p_norm__,
//==                                   int gkvec_reduced__,
//==                                   int mpi_rank__)
//== {
//==     dim3 grid_t(64);
//==     dim3 grid_b(num_blocks(num_gvec_loc__, grid_t.x), num_res_local__);
//== 
//==     compute_residuals_gpu_kernel <<<grid_b, grid_t>>>
//==     (
//==         num_gvec_loc__,
//==         eval__,
//==         hpsi__,
//==         opsi__,
//==         res__
//==     );
//== 
//==     grid_b = dim3(num_res_local__);
//== 
//==     compute_residuals_norm_gpu_kernel <<<grid_b, grid_t, grid_t.x * sizeof(double)>>>
//==     (
//==         num_gvec_loc__,
//==         res_idx__,
//==         res__,
//==         res_norm__,
//==         gkvec_reduced__,
//==         mpi_rank__
//==     );
//== 
//==     grid_b = dim3(num_blocks(num_gvec_loc__, grid_t.x), num_res_local__);
//== 
//==     apply_preconditioner_gpu_kernel <<<grid_b, grid_t>>>
//==     (
//==         num_gvec_loc__,
//==         res_idx__,
//==         eval__,
//==         h_diag__,
//==         o_diag__,
//==         res__
//==     );
//== 
//==     grid_b = dim3(num_res_local__);
//== 
//==     compute_residuals_norm_gpu_kernel <<<grid_b, grid_t, grid_t.x * sizeof(double)>>>
//==     (
//==         num_gvec_loc__,
//==         res_idx__,
//==         res__,
//==         p_norm__,
//==         gkvec_reduced__,
//==         mpi_rank__
//==     );
//== }

extern "C" void compute_residuals_gpu(acc_complex_double_t* hpsi__,
                                      acc_complex_double_t* opsi__,
                                      acc_complex_double_t* res__,
                                      int num_rows_loc__,
                                      int num_bands__,
                                      double* eval__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_rows_loc__, grid_t.x), num_bands__);

    accLaunchKernel((compute_residuals_gpu_kernel), dim3(grid_b), dim3(grid_t), 0, 0, 
        num_rows_loc__,
        eval__,
        hpsi__,
        opsi__,
        res__
    );
}

__global__ void add_square_sum_gpu_kernel
(
    int num_rows_loc__,
    acc_complex_double_t const* wf__,
    int reduced__,
    int mpi_rank__,
    double* result__
)
{
    int N = num_blocks(num_rows_loc__, blockDim.x);

    ACC_DYNAMIC_SHARED( char, sdata_ptr)
    double* sdata = (double*)&sdata_ptr[0];

    sdata[threadIdx.x] = 0.0;

    for (int n = 0; n < N; n++) {
        int j = n * blockDim.x + threadIdx.x;
        if (j < num_rows_loc__) {
            int k = array2D_offset(j, blockIdx.x, num_rows_loc__);
            sdata[threadIdx.x] += (wf__[k].x * wf__[k].x + wf__[k].y * wf__[k].y);
        }
    }
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) {
        if (threadIdx.x % (2 * s) == 0) {
            sdata[threadIdx.x] = sdata[threadIdx.x] + sdata[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        if (!reduced__) {
            result__[blockIdx.x] += sdata[0];
        } else {
            if (mpi_rank__ == 0) {
                double x = wf__[array2D_offset(0, blockIdx.x, num_rows_loc__)].x;
                result__[blockIdx.x] += (2 * sdata[0] - x * x);
            }
            else {
                result__[blockIdx.x] += 2 * sdata[0];
            }
        }
    }
}

extern "C" void add_square_sum_gpu(acc_complex_double_t* wf__,
                                   int num_rows_loc__,
                                   int nwf__,
                                   int reduced__,
                                   int mpi_rank__,
                                   double* result__)
{
    dim3 grid_t(64);
    dim3 grid_b(nwf__);

    accLaunchKernel((add_square_sum_gpu_kernel), dim3(grid_b), dim3(grid_t), grid_t.x * sizeof(double), 0, 
        num_rows_loc__,
        wf__,
        reduced__,
        mpi_rank__,
        result__
    );
}

__global__ void apply_preconditioner_gpu_kernel(int const num_rows_loc__,
                                                double const* eval__,
                                                double const* h_diag__,
                                                double const* o_diag__,
                                                acc_complex_double_t* res__)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int ibnd = blockIdx.y;

    if (j < num_rows_loc__) {
        double p = (h_diag__[j] - eval__[ibnd] * o_diag__[j]);
        p = 0.5 * (1 + p + sqrt(1.0 + (p - 1) * (p - 1)));
        int k = array2D_offset(j, ibnd, num_rows_loc__);
        res__[k] = make_accDoubleComplex(res__[k].x / p, res__[k].y / p);
    }
}

extern "C" void apply_preconditioner_gpu(acc_complex_double_t* res__,
                                         int num_rows_loc__,
                                         int num_bands__,
                                         double* eval__,
                                         const double* h_diag__,
                                         const double* o_diag__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_rows_loc__, grid_t.x), num_bands__);

    accLaunchKernel((apply_preconditioner_gpu_kernel), dim3(grid_b), dim3(grid_t), 0, 0, num_rows_loc__, eval__, h_diag__, o_diag__, res__);
}

__global__ void make_real_g0_gpu_kernel(acc_complex_double_t* res__,
                                        int              ld__)
{
    acc_complex_double_t z = res__[array2D_offset(0, blockIdx.x, ld__)];
    if (threadIdx.x == 0) {
        res__[array2D_offset(0, blockIdx.x, ld__)] = make_accDoubleComplex(z.x, 0);
    }
}

extern "C" void make_real_g0_gpu(acc_complex_double_t* res__,
                                 int              ld__,
                                 int              n__)
{
    dim3 grid_t(32);
    dim3 grid_b(n__);

    accLaunchKernel((make_real_g0_gpu_kernel), dim3(grid_b), dim3(grid_t), 0, 0, res__, ld__);
}






















